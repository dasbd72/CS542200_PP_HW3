#include "hip/hip_runtime.h"
#include <immintrin.h>
#include <omp.h>
#include <pthread.h>

#include <cassert>
#include <cmath>
#include <cstdio>
#include <iostream>
#include <queue>
#include <thread>
#include <utility>
#include <vector>

#ifdef DEBUG
#define DEBUG_PRINT(fmt, args...) fprintf(stderr, fmt, ##args);
#define DEBUG_MSG(str) std::cout << str << "\n";
#else
#define DEBUG_PRINT(fmt, args...)
#define DEBUG_MSG(str)
#endif  // DEBUG

#ifdef TIMING
#include <ctime>
#define TIMING_START(arg)          \
    struct timespec __start_##arg; \
    clock_gettime(CLOCK_MONOTONIC, &__start_##arg);
#define TIMING_END(arg)                                                                       \
    {                                                                                         \
        struct timespec __temp_##arg, __end_##arg;                                            \
        double __duration_##arg;                                                              \
        clock_gettime(CLOCK_MONOTONIC, &__end_##arg);                                         \
        if ((__end_##arg.tv_nsec - __start_##arg.tv_nsec) < 0) {                              \
            __temp_##arg.tv_sec = __end_##arg.tv_sec - __start_##arg.tv_sec - 1;              \
            __temp_##arg.tv_nsec = 1000000000 + __end_##arg.tv_nsec - __start_##arg.tv_nsec;  \
        } else {                                                                              \
            __temp_##arg.tv_sec = __end_##arg.tv_sec - __start_##arg.tv_sec;                  \
            __temp_##arg.tv_nsec = __end_##arg.tv_nsec - __start_##arg.tv_nsec;               \
        }                                                                                     \
        __duration_##arg = __temp_##arg.tv_sec + (double)__temp_##arg.tv_nsec / 1000000000.0; \
        printf("%s took %lfs.\n", #arg, __duration_##arg);                                    \
    }
#else
#define TIMING_START(arg)
#define TIMING_END(arg)
#endif  // TIMING

#define block_size 32
const int INF = ((1 << 30) - 1);

struct edge_t {
    int src;
    int dst;
    int w;
};

int blk_idx(int r, int c, int nblocks);

void proc(int *blk_dist, int s_i, int e_i, int s_j, int e_j, int k, int nblocks, int ncpus);

__global__ void proc_1_glob(int *blk_dist, int k, int nblocks);
__global__ void proc_2_glob(int *blk_dist, int s, int k, int nblocks);
__global__ void proc_3_glob(int *blk_dist, int s_i, int s_j, int k, int nblocks);

int main(int argc, char **argv) {
    assert(argc == 3);

    char *input_filename = argv[1];
    char *output_filename = argv[2];
    FILE *input_file;
    FILE *output_file;
    int ncpus = omp_get_max_threads();
    int V, E;
    edge_t *edge;
    int *dist;
    int VP;
    int nblocks;
    int *blk_dist;
    int *blk_dist_dev;

    TIMING_START(hw3_1);

    /* input */
    TIMING_START(input);
    input_file = fopen(input_filename, "rb");
    assert(input_file);
    fread(&V, sizeof(int), 1, input_file);
    fread(&E, sizeof(int), 1, input_file);
    edge = (edge_t *)malloc(sizeof(edge_t) * E);
    fread(edge, sizeof(edge_t), E, input_file);
    dist = (int *)malloc(sizeof(int) * V * V);
    fclose(input_file);
    DEBUG_PRINT("vertices: %d\nedges: %d\n", V, E);
    TIMING_END(input);

    /* calculate */
    TIMING_START(calculate);
    nblocks = (int)ceilf(float(V) / block_size);
    VP = nblocks * block_size;
    blk_dist = (int *)malloc(sizeof(int) * VP * VP);

    for (int i = 0; i < VP; i++) {
        for (int j = 0; j < VP; j++) {
            if (i == j)
                blk_dist[blk_idx(i, j, nblocks)] = 0;
            else
                blk_dist[blk_idx(i, j, nblocks)] = INF;
        }
    }

    for (int i = 0; i < E; i++) {
        blk_dist[blk_idx(edge[i].src, edge[i].dst, nblocks)] = edge[i].w;
    }

    hipHostRegister(blk_dist, sizeof(int) * VP * VP, hipHostRegisterDefault);
    hipMalloc(&blk_dist_dev, sizeof(int) * VP * VP);
    hipMemcpy(blk_dist_dev, blk_dist, sizeof(int) * VP * VP, hipMemcpyHostToDevice);

    dim3 blk(block_size, block_size);
    for (int k = 0, nk = nblocks - 1; k < nblocks; k++, nk--) {
        /* Phase 1 */
        proc_1_glob<<<1, blk>>>(blk_dist_dev, k, nblocks);
        /* Phase 2 */
        if (k)
            proc_2_glob<<<k, blk>>>(blk_dist_dev, 0, k, nblocks);
        if (nk)
            proc_2_glob<<<nk, blk>>>(blk_dist_dev, k + 1, k, nblocks);
        /* Phase 3 */
        if (k)
            proc_3_glob<<<dim3(k, k), blk>>>(blk_dist_dev, 0, 0, k, nblocks);
        if (k && nk)
            proc_3_glob<<<dim3(nk, k), blk>>>(blk_dist_dev, 0, k + 1, k, nblocks);
        if (k && nk)
            proc_3_glob<<<dim3(k, nk), blk>>>(blk_dist_dev, k + 1, 0, k, nblocks);
        if (nk)
            proc_3_glob<<<dim3(nk, nk), blk>>>(blk_dist_dev, k + 1, k + 1, k, nblocks);
    }

    hipMemcpy(blk_dist, blk_dist_dev, sizeof(int) * VP * VP, hipMemcpyDeviceToHost);

    /* Copy result to dist */
    for (int i = 0; i < V; i++) {
        for (int j = 0; j < V; j++) {
            dist[i * V + j] = min(blk_dist[blk_idx(i, j, nblocks)], INF);
        }
    }

    TIMING_END(calculate);

    /* output */
    TIMING_START(output);
    output_file = fopen(output_filename, "w");
    assert(output_file);
    fwrite(dist, sizeof(int), V * V, output_file);
    fclose(output_file);
    TIMING_END(output);
    TIMING_END(hw3_1);

    /* finalize */
    free(edge);
    free(dist);
    free(blk_dist);
    hipFree(blk_dist_dev);
    return 0;
}

int blk_idx(int r, int c, int nblocks) {
    return ((r / block_size) * nblocks + (c / block_size)) * block_size * block_size + (r % block_size) * block_size + (c % block_size);
}

void proc(int *blk_dist, int s_i, int e_i, int s_j, int e_j, int k, int nblocks, int ncpus) {
#pragma omp parallel for num_threads(ncpus) schedule(static) default(shared) collapse(2)
    for (int i = s_i; i < e_i; i++) {
        for (int j = s_j; j < e_j; j++) {
            int *ik_ptr = blk_dist + (i * nblocks + k) * block_size * block_size;
            int *ij_ptr = blk_dist + (i * nblocks + j) * block_size * block_size;
            int *kj_ptr = blk_dist + (k * nblocks + j) * block_size * block_size;
            for (int b = 0; b < block_size; b++) {
                for (int r = 0; r < block_size; r++) {
#pragma omp simd
                    for (int c = 0; c < block_size; c++) {
                        ij_ptr[r * block_size + c] = std::min(ij_ptr[r * block_size + c], ik_ptr[r * block_size + b] + kj_ptr[b * block_size + c]);
                    }
                }
            }
        }
    }
}

__global__ void proc_1_glob(int *blk_dist, int k, int nblocks) {
    __shared__ int k_k_sm[block_size][block_size];

    int r = threadIdx.y;
    int c = threadIdx.x;
    int *k_k_ptr = blk_dist + (k * nblocks + k) * (block_size * block_size);
    int tmp;

    k_k_sm[r][c] = k_k_ptr[r * block_size + c];
    __syncthreads();

#pragma unroll 32
    for (int b = 0; b < block_size; b++) {
        tmp = k_k_sm[r][b] + k_k_sm[b][c];
        if (tmp < k_k_sm[r][c])
            k_k_sm[r][c] = tmp;
        __syncthreads();
    }
    k_k_ptr[r * block_size + c] = k_k_sm[r][c];
}
__global__ void proc_2_glob(int *blk_dist, int s, int k, int nblocks) {
    __shared__ int i_k_sm[block_size][block_size];
    __shared__ int k_j_sm[block_size][block_size];
    __shared__ int k_k_sm[block_size][block_size];

    int i = s + blockIdx.x;
    int j = s + blockIdx.x;
    int r = threadIdx.y;
    int c = threadIdx.x;
    int *i_k_ptr = blk_dist + (i * nblocks + k) * (block_size * block_size);
    int *k_j_ptr = blk_dist + (k * nblocks + j) * (block_size * block_size);
    int *k_k_ptr = blk_dist + (k * nblocks + k) * (block_size * block_size);
    int tmp;

    i_k_sm[r][c] = i_k_ptr[r * block_size + c];
    k_j_sm[r][c] = k_j_ptr[r * block_size + c];
    k_k_sm[r][c] = k_k_ptr[r * block_size + c];
    __syncthreads();

#pragma unroll 32
    for (int b = 0; b < block_size; b++) {
        i_k_sm[r][c] = min(i_k_sm[r][c], i_k_sm[r][b] + k_k_sm[b][c]);
        k_j_sm[r][c] = min(k_j_sm[r][c], k_k_sm[r][b] + k_j_sm[b][c]);
        __syncthreads();
    }
    i_k_ptr[r * block_size + c] = i_k_sm[r][c];
    k_j_ptr[r * block_size + c] = k_j_sm[r][c];
}
__global__ void proc_3_glob(int *blk_dist, int s_i, int s_j, int k, int nblocks) {
    __shared__ int i_k_sm[block_size][block_size];
    __shared__ int k_j_sm[block_size][block_size];

    int i = s_i + blockIdx.y;
    int j = s_j + blockIdx.x;
    int r = threadIdx.y;
    int c = threadIdx.x;
    int *i_k_ptr = blk_dist + (i * nblocks + k) * (block_size * block_size);
    int *i_j_ptr = blk_dist + (i * nblocks + j) * (block_size * block_size);
    int *k_j_ptr = blk_dist + (k * nblocks + j) * (block_size * block_size);
    int loc, tmp;

    i_k_sm[r][c] = i_k_ptr[r * block_size + c];
    k_j_sm[r][c] = k_j_ptr[r * block_size + c];
    __syncthreads();
    loc = i_j_ptr[r * block_size + c];

#pragma unroll 32
    for (int b = 0; b < block_size; b++) {
        tmp = i_k_sm[r][b] + k_j_sm[b][c];
        if (tmp < loc)
            loc = tmp;
    }
    i_j_ptr[r * block_size + c] = loc;
}